#include "hip/hip_runtime.h"
//
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "Types.h"
#include <cuda/helpers.h>
#include <cuda/random.h>

#include <sutil/vec_math.h>

const float  PI = 3.1415927f;

extern "C" {
	__constant__ Params params;
}

struct TraceData
{
	float3 color;
	float3 origin;
	float3 normal;
	float3 throughput;
	int depth;
	unsigned int seed;
};


static __forceinline__ __device__ void* unpackPointer(unsigned int i0, unsigned int i1)
{
	const unsigned long long uptr = static_cast<unsigned long long>(i0) << 32 | i1;
	void*           ptr = reinterpret_cast<void*>(uptr);
	return ptr;
}


static __forceinline__ __device__ void  packPointer(void* ptr, unsigned int& i0, unsigned int& i1)
{
	const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
	i0 = uptr >> 32;
	i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ TraceData* getTraceData()
{
	const unsigned int u0 = optixGetPayload_0();
	const unsigned int u1 = optixGetPayload_1();
	return reinterpret_cast<TraceData*>(unpackPointer(u0, u1));
}


static __forceinline__ __device__ void setPayload(float3 p)
{
	optixSetPayload_0(float_as_int(p.x));
	optixSetPayload_1(float_as_int(p.y));
	optixSetPayload_2(float_as_int(p.z));
}


static __forceinline__ __device__ void computeRay(uint3 idx, float2 subpixel_jitter, uint3 dim, float3& origin, float3& direction)
{
	const float3 U = params.cam_u;
	const float3 V = params.cam_v;
	const float3 W = params.cam_w;
	const float2 d = 2.0f * make_float2(
		(static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(dim.x),
		(static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(dim.y)
	) - 1.0f;

	origin = params.cam_eye;
	direction = normalize(d.x * U - d.y * V + W);
}

static __forceinline__ __device__ void copy(float t[16], float (& t2)[16])
{
	for (int i = 0; i < 16; i++)
	{
		t2[i] = t[i];
	}
}

static __forceinline__ __device__ float3 transform(float3 ray, float(&t)[16])
{
	float3 ray2;
	ray2.x = (t[0] * ray.x) + (t[1] * ray.y) + (t[2] * ray.z) + t[3];
	ray2.y = (t[4] * ray.x) + (t[5] * ray.y) + (t[6] * ray.z) + t[7];
	ray2.z = (t[8] * ray.x) + (t[9] * ray.y) + (t[10] * ray.z) + t[11];
	return ray2;
}


static __forceinline__ __device__ float dot0(float3 f1, float3 f2)
{
	return fmax(dot(f1, f2), 0.0f);
}


static __forceinline__ __device__ void transpose(float (& t)[16])
{
	float t1[16];
	copy(t, t1);

	t[1] = t1[4];
	t[2] = t1[8];
	t[3] = t1[12];
	t[4] = t1[1];
	t[6] = t1[9];
	t[7] = t1[13];
	t[8] = t1[2];
	t[9] = t1[6];
	t[11] = t1[14];
	t[12] = t1[3];
	t[13] = t1[7];
	t[14] = t1[11];
}


static __forceinline__ __device__ bool traceOcclusion(
	OptixTraversableHandle handle,
	float3                 ray_origin,
	float3                 ray_direction,
	float                  tmin,
	float                  tmax)
{
	unsigned int occluded = 0u;
	optixTrace(
		handle,
		ray_origin,
		ray_direction,
		tmin,
		tmax,
		0.0f,                    // rayTime
		OptixVisibilityMask(255),
		OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
		RAY_TYPE_OCCLUSION,      // SBT offset
		RAY_TYPE_COUNT,          // SBT stride
		RAY_TYPE_OCCLUSION,      // missSBTIndex
		occluded);
	return occluded;
}



static __forceinline__ __device__ float3 traceRadiance(
	OptixTraversableHandle handle,
	float3                 ray_origin,
	float3                 ray_direction,
	int					   depth,
	unsigned int&		   seed,
	float3&				   throughput)
{
	TraceData td;
	td.depth = depth;
	td.seed = seed;
	td.throughput = throughput;

	unsigned int u0, u1;
	packPointer(&td, u0, u1);

	optixTrace(
		handle,
		ray_origin,
		ray_direction,
		0.0f,					// tmin
		1e16f,					// tmax
		0.0f,                    // rayTime
		OptixVisibilityMask(255),
		OPTIX_RAY_FLAG_NONE,
		RAY_TYPE_RADIANCE,      // SBT offset
		RAY_TYPE_COUNT,         // SBT stride
		RAY_TYPE_RADIANCE,      // missSBTIndex
		u0, u1);
	
	seed = td.seed;
	throughput = td.throughput;
	return td.color;
}




static __forceinline__ __device__ uchar4 make_color_no_gamma(float3 c)
{
	return make_uchar4(quantizeUnsigned8Bits(c.x), quantizeUnsigned8Bits(c.y), quantizeUnsigned8Bits(c.z), 255u);
}



extern "C"
__global__ void __raygen__rg()
{
	// Lookup our location within the launch grid
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();
	
	// First ray goes through the center of the pixel
	float2 subpixel_jitter = make_float2(0.5f);

	unsigned int seed = tea<4>(idx.y*params.image_width + idx.x, 0);

	// Map our launch idx to a screen location and create a ray from the camera
	// location through the screen
	float3 ray_origin, ray_direction;
	float3 throughput = make_float3(1);

	// Iterate over all samples-per-pixel
	float3 result = make_float3(0);
	for (int p = 0; p < params.spp; p++)
	{
		computeRay(idx, subpixel_jitter, dim, ray_origin, ray_direction);
		result += traceRadiance(params.handle, ray_origin, ray_direction, 0, seed, throughput);

		// Subsequent rays go through random point in pixel
		subpixel_jitter = make_float2(rnd(seed), rnd(seed));
	}

	// Take the average of the samples for this pixel
	result /= params.spp;

	// Record results in our output raster
	params.image[idx.y * params.image_width + idx.x] = make_color_no_gamma(result);
}



extern "C" __global__ void __miss__ms() 
{
	MissData* miss_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());

	TraceData* td = getTraceData();
	td->color = miss_data->bg_color;
}

float3 rayTracerShade(float3 N, HitGroupData* hit_data) 
{
	const float3 orig = optixGetWorldRayOrigin();
	const float3 dir = optixGetWorldRayDirection();
	const float  t = optixGetRayTmax();
	const float3 P = orig + t * dir; // hit point

	DLight* dl = (DLight*)params.lights;

	float3 c = make_float3(0);
	c += hit_data->ambient + hit_data->emission;

	for (int i = 0; i < params.light_count; i++)
	{
		DLight light = dl[i];
		float3 L = make_float3(0);

		float intensity = 1.0f;
		float LDist;
		if (light.type == 0) // point
		{
			L = normalize(light.position - P);
			LDist = length(light.position - P);
			intensity = 1.0f / (light.atten0 + light.atten1 * LDist + light.atten2 * LDist * LDist);
		}
		else // directional
		{
			L = normalize(light.position);
			LDist = 1e16f;
		}

		//float LdotN = max( dot(L, N), 0.0f);
		float LdotN = dot(L, N);
		LdotN = LdotN < 0 ? 0 : LdotN;
		float3 lambert = hit_data->diffuse * LdotN;

		float3 H = normalize(L + (normalize(orig - P)));
		float HdotN = dot(H, N);
		HdotN = (HdotN < 0) ? 0 : HdotN;
		float3 phong = make_float3(0);
		if (hit_data->shininess > 0)
			phong = hit_data->specular * pow(HdotN, hit_data->shininess);


		// calculate shadow
		float V = 1.0f;
		float3 occlusionOrig = P + N * EPSILON;
		float3 occlusionDir = L;

		const bool occluded = traceOcclusion(
			params.handle, occlusionOrig, occlusionDir,
			0.0001f, LDist);

		if (occluded)
		{
			V = 0;
		}

		c += V * intensity * light.color * (lambert + phong);
	}

	TraceData* td = getTraceData();


	// specularity
	float s = hit_data->specular.x + hit_data->specular.y + hit_data->specular.x;
	if (s > 0 && td->depth < params.depth)
	{
		td->origin = P;

		float3 r = normalize(orig - P);
		float3 refl = N * 2.0f * dot(r, N) - r;
		refl = normalize(refl);

		float3 reflOrigin = P + N * EPSILON;

		// trace reflection
		float3 reflColor = traceRadiance(params.handle, reflOrigin, refl, td->depth + 1, td->seed, td->throughput);
		c += hit_data->specular * reflColor;
	}

	return c;
}

float3 analyticDirectShade(float3 N, HitGroupData* hit_data) 
{
	const float3 orig = optixGetWorldRayOrigin();
	const float3 dir = optixGetWorldRayDirection();
	const float  t = optixGetRayTmax();
	const float3 P = orig + t * dir; // hit point

	DQuadLight* dql = (DQuadLight*)params.quadLights;
	float3 c = make_float3(0);

	c += hit_data->ambient + hit_data->emission;

	for (int j = 0; j < params.quad_light_count; j++)
	{
		DQuadLight ql = dql[j];
		const int vertexCount = 4;
		float3 verticies[4];
		verticies[0] = ql.a;
		verticies[1] = ql.a + ql.ab;
		verticies[2] = ql.a + ql.ab + ql.ac;
		verticies[3] = ql.a + ql.ac;



		float3 irradiance = make_float3(0);

		for (int i = 0; i < vertexCount; i++) {
			float3 v1 = verticies[i];
			float3 v2 = verticies[(i + 1) % vertexCount];
			float theta = acosf(dot(normalize(v1 - P), normalize(v2 - P)));
			float3 gamma = normalize(cross(v1 - P, v2 - P));

			irradiance += theta * gamma;
		}

		irradiance *= 0.5f;

		c += (hit_data->diffuse / PI) * ql.intensity * dot(irradiance, N);
	}

	return c;
}

float3 brdf(float3 omegaI, float3 dir, float3 N, float3 kd, float3 ks, float s) 
{
	// reflection vector of sample
	float3 refl = normalize((2.0f * dot0(-dir, N) * N) + dir);

	float3 result = kd / PI;						// Lambert shading

	float rDotWi = fmax(0, dot0(refl, omegaI));		// Specular shading
	if (length(ks) > 0 && rDotWi > 0)
		result += ks * ((s + 2.0f) / (2.0f*PI)) * pow(rDotWi, s);

	return result;
}

float3 directShade(float3 N, HitGroupData* hit_data) 
{
	const int light_samples = params.light_samples;
	const int strat_grid = params.light_stratify ? sqrtf(light_samples) : 1;
	
	TraceData* td = getTraceData();

	const float3 orig = optixGetWorldRayOrigin();
	const float3 dir = optixGetWorldRayDirection();
	const float  t = optixGetRayTmax();
	const float3 P = orig + t * dir; // hit point

	DQuadLight* dql = (DQuadLight*)params.quadLights;
	float3 fc = make_float3(0);

	if (params.nee == 0)
		fc += hit_data->ambient + hit_data->emission;

	for (int j = 0; j < params.quad_light_count; j++) 
	{
		float3 col = make_float3(0);
		DQuadLight ql = dql[j];

		float3 a = ql.a;							// verticies of quad light
		float3 b = ql.a + ql.ab;
		float3 c = ql.a + ql.ac;
		float3 nl = normalize(cross(c - a, b - a));	// surface normal of the area light

		float A = length(ql.ab) * length(ql.ac);	// area of parallelogram
		//float A = length(cross(ql.ab, ql.ac));

		for (int k = 0; k < light_samples; k++) {

			float u1 = rnd(td->seed);
			float u2 = rnd(td->seed);

			int si, sj;
			if (params.light_stratify)
			{
				si = k / strat_grid;						// Stratified grid cell i
				sj = k % strat_grid;						// Stratified grid cell j
			}
			else {
				si = sj = 0;
			}
			
			float3 x1 = ql.a								// sampled point in light source
				+ ((sj + u1)/strat_grid) * ql.ab
				+ ((si + u2)/strat_grid) * ql.ac;		

			float3 omegaI = normalize(x1 - P);				// direction vector from hit point to light sample
			float R = length(x1 - P);						// distance from hit point to light sample
			float nDotWi = dot0(N, omegaI);					// Cosine component
			float LnDotWi = dot0(-nl, omegaI);				// differential omegaI
			
			// Visibility of sample
			const bool occluded = traceOcclusion(params.handle, P, omegaI,
				0.0001f, R);
			float V = occluded ? 0 : 1;

			// BRDF
			float3 f = brdf(omegaI, dir, N, hit_data->diffuse, hit_data->specular, hit_data->shininess);

			col += V * f * nDotWi * LnDotWi / (R * R);	// Put it all together
		}

		fc += (col * ql.intensity * A) / light_samples;
	}

	return fc;
}

float3 pathTraceShade(float3 N, HitGroupData* hit_data)
{
	TraceData* td = getTraceData();

	// Exit on maximum recusion depth
	int max_depth = params.depth;
	if (params.nee == 1)
		max_depth--;

	if (td->depth >= max_depth)
	{
		if (params.nee == 1)
			return make_float3(0);
		else
			return hit_data->emission;
	}

	// Exit if we intersect the light source
	if (hit_data->primativeType == QUADLIGHT)
	{
		if (params.nee == 1)
			return make_float3(0);
		else
			return hit_data->emission;
	}

	// 
	// If using Russian Roulette terminate based on probability q
	//
	float q = 1;						// termination probability 
	float rrBoost = 1.0f;
	if (params.russian_roulette == 1)
	{
		// choose paths with lower throughput to terminate more frequently 
		q = 1.0f - fmin(fmax(fmax(td->throughput.x, td->throughput.y), td->throughput.z), 1.0f);
		float p = rnd(td->seed);
		if (p < q)
		{
			return make_float3(0);		// terminate
		}
		else
		{
			if(q < 1.0f)
				rrBoost = 1.0f / (1.0f - q);	// boost paths that are not terminated 
		}
	}

	const float3 orig = optixGetWorldRayOrigin();
	const float3 dir = optixGetWorldRayDirection();
	const float  t = optixGetRayTmax();
	const float3 P = orig + t * dir; // hit point

	// randomly generate hemisphere sample
	float psi1 = rnd(td->seed);
	float psi2 = rnd(td->seed);

	float theta = acos(clamp(psi1, 0.0f, 1.0f));		// random number between pi/2 and 0
	float phi = 2.0f * PI * psi2;						// random number between 0 and 2*pi

	// calcualte sample in cartesian coordinates 
	float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));

	// rotate sample to be centered about normal N
	float3 a = make_float3(0, 1, 0);
	if (dot(a, N) > 0.9)
		a = make_float3(1, 0, 0);
	float3 w = normalize(N);
	float3 u = normalize(cross(a, w));
	float3 v = cross(w, u);

	// direction vector to next sample
	float3 omegaI = normalize(s.x * u + s.y * v + s.z * w);

	// BRDF
	float3 f = brdf(omegaI, dir, N, hit_data->diffuse, hit_data->specular, hit_data->shininess);

	float nDotWi = dot0(N, omegaI);				// Cosine component

	// Apply throughput for next hop in path
	td->throughput *= 2.0f * PI * f * nDotWi;

	// Recursively sample next color along path
	float3 nextColor = traceRadiance(params.handle, P + EPSILON * N, omegaI, td->depth + 1, td->seed, td->throughput);
	//nextColor = clamp(nextColor, 0.0f, 1.0f);

	float3 Lo = 2.0f * PI * f * nextColor * nDotWi * rrBoost;
	
	//return clamp(Lo, 0, 6);
	return Lo;
}


void shade(float3 N, HitGroupData* hit_data)
{
	TraceData* td = getTraceData();

	if (params.integrator == RAYTRACER) {
		td->color = rayTracerShade(N, hit_data);
	}
	else if (params.integrator == ANALYTICDIRECT) {
		td->color = analyticDirectShade(N, hit_data);
	}
	else if (params.integrator == DIRECT) {
		td->color = directShade(N, hit_data);
	}
	else if (params.integrator == PATHTRACER) {
		if (params.nee == 1) {
			td->color = make_float3(0);

			if(td->depth == 0)
				td->color += hit_data->ambient + hit_data->emission;

			td->color += directShade(N, hit_data);
			td->color += pathTraceShade(N, hit_data);
		}
		else
		{
			// NEE off, do indirect lighting only 
			td->color = pathTraceShade(N, hit_data);
		}
	}
}


extern "C" __global__ void __closesthit__primative()
{
	// When built-in triangle intersection is used, a number of fundamental
	// attributes are provided by the OptiX API, indlucing barycentric coordinates.
	//const float2 barycentrics = optixGetTriangleBarycentrics();


	HitGroupData* hit_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());

	const float3 normal =
		make_float3(
			int_as_float(optixGetAttribute_0()),
			int_as_float(optixGetAttribute_1()),
			int_as_float(optixGetAttribute_2())
		);

	shade(normal, hit_data);

}



extern "C" __global__ void __closesthit__occlusion()
{
	optixSetPayload_0(static_cast<unsigned int>(true));
}


bool intersect_triangle(HitGroupData* hg_data, float3 orig, float3 dir, float3 &normal, float& t)
{
	float3 a = hg_data->verticies[0];
	float3 b = hg_data->verticies[1];
	float3 c = hg_data->verticies[2];
	normal = normalize(cross(c - a, b - a));

	float dirDotN(dot(dir, normal));

	// No ray-plane intersection (orthogonal)
	if (dirDotN == 0)
		return false;


	// Calculate distance along ray to intersection point
	t = (dot(a, normal) - dot(orig, normal)) / dirDotN;

	// Intersections from behind origin are invalid
	if (t < 0)
		return false;

	float3 P = orig + dir * t;
	

	// Intersection with the triangle's plane, determine if it's inside or outside
	float3 beta = cross(P - a, b - a);
	if (dot(normal, beta) < 0)
		return false;

	float3 gamma = cross(P - b, c - b);
	if (dot(normal, gamma) < 0)
		return false;

	float3 alpha = cross(P - c, a - c);
	if (dot(normal, alpha) < 0)
		return false;

	// TODO: This I don't fully understand.  But I know that the normal is facing the wrong direction
	// if it is not reversed here.   
	normal *= -1;

	return true;
}

bool intersect_sphere(HitGroupData* hg_data, float3 orig, float3 dir, float3 &normal, float& t)
{

	float3 C = hg_data->sphere.center;
	float r = hg_data->sphere.radius;

	float3 p0MinusC = orig - C;
	float a = dot(dir, dir);
	float b = dot(dir * 2, p0MinusC);
	float c = dot(p0MinusC, p0MinusC) - r * r;

	float discriminant = b * b - 4 * a*c;
	if (discriminant > 0.0f) {
		int hit = 1;
		float sDiscriminant = sqrt(discriminant);
		float t1 = (-b + sDiscriminant) / (2.0f*a);
		float t2 = (-b - sDiscriminant) / (2.0f*a);

		if (t1 > 0.0f && t2 > 0.0f)
			t = t1 < t2 ? t1 : t2; // min(t1, t2);
		else if (t1 > 0.0f)
			t = t1;
		else if (t2 > 0.0f)
			t = t2;
		else
			hit = 0;

		if (hit > 0) {
			float3 hitPosition = orig + dir * t;
			normal = normalize(hitPosition - C);
			return true;
		}
	}

	return false;
}


void printRay(float3 orig, float3 dir)
{
	printf("origin: %f, %f, %f | dir: %f, %f, %f \n", orig.x, orig.y, orig.z, dir.x, dir.y, dir.z);
}


extern "C" __global__ void __intersection__primative()
{
	HitGroupData* hg_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());
	float3 orig = optixGetObjectRayOrigin();
	float3 dir = optixGetObjectRayDirection();
	unsigned int flags = optixGetRayFlags();

	// apply inverse transform
	orig = transform(orig, hg_data->inverseTransform);
	dir = transform(dir, hg_data->inverseWithoutTranslate);
	

	float3 normal;
	float t;
	bool hit = false;


	if(hg_data->primativeType == SPHERE) 
	{
		hit = intersect_sphere(hg_data, orig, dir, normal, t);
	}
	else if(hg_data->primativeType == TRIANGLE)
	{
		hit = intersect_triangle(hg_data, orig, dir, normal, t);
	}
	// Quad lights should not participate in occlusion checks.
	// Assume that we are doing an occlusion trace based on flags
	else if (hg_data->primativeType == QUADLIGHT && flags != OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT)
	{
		hit = intersect_triangle(hg_data, orig, dir, normal, t);
	}

	if(hit) 
	{
		// and apply the transpose inverse matrix to the normal
		float transp[16];
		copy(hg_data->inverseTransform, transp);
		transpose(transp);
		normal = transform(normal, transp);
		normal = normalize(normal);

		optixReportIntersection(
			t,      // t hit
			0,          // user hit kind
			float_as_int(normal.x),
			float_as_int(normal.y),
			float_as_int(normal.z)
		);
	}
}


